#include "hip/hip_runtime.h"
#include "dtm.h"
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <math.h>

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

//definition of constant vector depth to world
__constant__ double  depthToWorld[] = {1.0 / 5.9421434211923247e+02, 1.0 / 5.9104053696870778e+02, 3.3930780975300314e+02, 2.4273913761751615e+02};

__constant__ double  worldToColor[] = {5.2921508098293293e+02, 5.2556393630057437e+02, 3.2894272028759258e+02, 2.6748068171871557e+02};


inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

/*__device__ void
transposeMat(uchar matIn[][], uchar matOut[][])
{
	uint16 tx(threadIdx.x);
	uint16 ty(threadIdx.y);
	matOut[tx][ty] = matIn[ty][tx];
}

__device__ void
transposeMat(float matIn[][], float matOut[][])
{
	uint16 tx(threadIdx.x);
	uint16 ty(threadIdx.y);
	matOut[tx][ty] = matIn[ty][tx];
}*/

__device__ void
transformByPoint(SWorldPoint*** inPoints, const float* trMat, SWorldPoint*** transformedPoints)
{
	uint16 tx(threadIdx.x);
	uint16 ty(threadIdx.y);
	float tmpPoint[] = {inPoints[ty][tx]->x,inPoints[ty][tx]->y,inPoints[ty][tx]->z};

	float w(tmpPoint[0] * trMat[3] + tmpPoint[1] * trMat[7] + tmpPoint[2] * trMat[11] + trMat[15]);

	if (w)
	{
		float invW = 1.0f / w;
		transformedPoints[ty][tx]->x = (tmpPoint[0] * trMat[0] + tmpPoint[1] * trMat[4] + tmpPoint[2] * trMat[8] + trMat[12]) * invW;
		transformedPoints[ty][tx]->y = (tmpPoint[0] * trMat[1] + tmpPoint[1] * trMat[5] + tmpPoint[2] * trMat[9] + trMat[13]) * invW;
		transformedPoints[ty][tx]->z = (tmpPoint[0] * trMat[2] + tmpPoint[1] * trMat[6] + tmpPoint[2] * trMat[10] + trMat[14]) * invW;
	}
	else
	{
		transformedPoints[ty][tx]->x = tmpPoint[0];
		transformedPoints[ty][tx]->y = tmpPoint[1];
		transformedPoints[ty][tx]->z = tmpPoint[2];
	}
}

__global__ void
dtmKernel(const uint16 *in, uint16 *out,const int minDist, const int maxDist,const int cols, const int rows)
{
	__shared__  float sharedIn[N][N];
	__shared__  float sharedTmp[N][N];
	__shared__  float sharedOut[N][N];

	unsigned int tx(threadIdx.x);
	unsigned int ty(threadIdx.y);
	unsigned int ind(blockIdx.x*blockDim.x + blockIdx.y*blockDim.y*cols + ty*cols + tx);
	//unsigned int tid(ty*blockDim.x + tx);
	

	sharedIn[ty][tx] = in[ind];
	sharedOut[ty][tx] = 0;
	sharedTmp[ty][tx] = 0.f;
	__syncthreads();

	if (sharedIn[ty][tx] < 2047)
	{
		sharedTmp[ty][tx] = static_cast<float>(1.0f / static_cast<float>(sharedIn[ty][tx]*-0.0030711016f + 3.330949516f));


		if ((sharedTmp[ty][tx] >= minDist) && (sharedTmp[ty][tx] <= maxDist))
		{
			sharedOut[ty][tx] = sharedIn[ty][tx];
		}
	}

	__syncthreads();

	//out[ind] = sharedIn[ty][tx];
	out[ind] = static_cast<uint16>(sharedOut[ty][tx]);
}


__global__ void
dtmKernelColor(const uint16 *in, const uchar *inCol ,uchar *out,const int minDist, const int maxDist,const int cols, const int rows)
{
	__shared__  uint16 sharedIn[N][N];
	__shared__  uchar sharedColor[N][N];
	__shared__  float sharedTmp[N][N];
	__shared__  uchar sharedOut[N][N];

	unsigned int tx(threadIdx.x);
	unsigned int ty(threadIdx.y);
	unsigned int ind(blockIdx.x*blockDim.x + blockIdx.y*blockDim.y*cols + ty*cols + tx);
	//unsigned int tid(ty*blockDim.x + tx);
	

	sharedIn[ty][tx] = in[ind];
	sharedOut[ty][tx] = 0;
	sharedTmp[ty][tx] = 0.f;
	sharedColor[ty][tx] = inCol[ind];
	__syncthreads();

	if (sharedIn[ty][tx] < 2047)
	{
		sharedTmp[ty][tx] = static_cast<float>(1.0 / static_cast<double>(sharedIn[ty][tx]*-0.0030711016 + 3.330949516));


		if ((sharedTmp[ty][tx] >= minDist) && (sharedTmp[ty][tx] <= maxDist))
		{
			sharedOut[ty][tx] = sharedColor[ty][tx];
		}
	}

	__syncthreads();

	out[ind] = sharedOut[ty][tx];
}

__global__ void
dtmKernelColorAlternative(const uint16 *in, uchar *inCol,const int minDist, const int maxDist,const int cols, const int rows)
{
	__shared__  float sharedIn[N][N];
	__shared__  float sharedColor[N][N];
	__shared__  float sharedTmp[N][N];

	unsigned int tx(threadIdx.x);
	unsigned int ty(threadIdx.y);
	unsigned int ind(blockIdx.x*blockDim.x + blockIdx.y*blockDim.y*cols + ty*cols + tx);
	//unsigned int tid(ty*blockDim.x + tx);


	sharedIn[ty][tx] = in[ind];
	sharedTmp[ty][tx] = 0.f;
	sharedColor[ty][tx] = inCol[ind];
	__syncthreads();

	if (sharedIn[ty][tx] < 2047)
	{
		sharedTmp[ty][tx] = static_cast<float>(1.0 / static_cast<float>(sharedIn[ty][tx]*-0.0030711016f + 3.330949516f));


		if ((sharedTmp[ty][tx] < minDist) || (sharedTmp[ty][tx] > maxDist))
		{
			//sharedColor[ty][tx] = 0;
			inCol[ind] = 0;
		}
	}

	__syncthreads();

	//inCol[ind] = static_cast<uchar>(sharedColor[ty][tx]);
}

__global__ void
depthToWorldColorKernel(const uint16 *in, const float *trMat ,SWorldPoint *out,const int minDist, const int maxDist,const int cols, const int rows)
{
	__shared__  uint16 sharedIn[N][N];
	__shared__  SWorldPoint* sharedOut[N][N];
	__shared__  float sharedTmp[N][N];

	unsigned int tx(threadIdx.x);
	unsigned int ty(threadIdx.y);
	unsigned int ind(blockIdx.x*blockDim.x + blockIdx.y*blockDim.y*cols + ty*cols + tx);
	//unsigned int tid(ty*blockDim.x + tx);


	sharedIn[ty][tx] = in[ind];
	sharedOut[ty][tx] = &out[ind];
	__syncthreads();

	//calculate the depth to meters
	if (sharedIn[ty][tx] < 2047)
	{
		sharedTmp[ty][tx] = static_cast<float>(1.0 / static_cast<double>(sharedIn[ty][tx]*-0.0030711016 + 3.330949516));


		if ((sharedTmp[ty][tx] < minDist) || (sharedTmp[ty][tx] > maxDist))
		{
			sharedTmp[ty][tx] = 0.f;
		}
	}

	__syncthreads();

	//calculate the real world data
	sharedOut[ty][tx]->x = (ty - depthToWorld[CX]) * sharedTmp[ty][tx] * depthToWorld[FX];
	sharedOut[ty][tx]->y = (tx - depthToWorld[CY]) * sharedTmp[ty][tx] * depthToWorld[FY];
	sharedOut[ty][tx]->z = sharedTmp[ty][tx];

	__syncthreads();

	//calc the transformed matrix
	transformByPoint(reinterpret_cast<SWorldPoint***>(sharedOut), trMat, reinterpret_cast<SWorldPoint***>(sharedOut));

	__syncthreads();

	//calc the colored pixels

	out[ind].x = round((sharedOut[ty][tx]->x * worldToColor[FX] * (1.f / sharedOut[ty][tx]->z)) + worldToColor[CX]);
	out[ind].y = round((sharedOut[ty][tx]->y * worldToColor[FY] * (1.f / sharedOut[ty][tx]->z)) + worldToColor[CY]);
}


float dtmGpu( uchar* h_in, uchar* h_out, const int rows, const int cols, const int minDist, const int maxDist)
{
	const int SIZE = rows * cols * sizeof(uint16);
	hipEvent_t start, stop;
	
	checkCudaErrors( hipEventCreate(&start) );
	checkCudaErrors( hipEventCreate(&stop) );

	// Allocate the device input image
	uchar *d_in = NULL;
	checkCudaErrors( hipMalloc((void **)&d_in, SIZE) );

	// Allocate the device output image
	uchar *d_out = NULL;
	checkCudaErrors( hipMalloc((void **)&d_out, SIZE) );

	// Copy the host input image  to the device memory
	checkCudaErrors( hipMemcpy(d_in, h_in, SIZE, hipMemcpyHostToDevice) );

	checkCudaErrors( hipEventRecord(start, NULL) );

	// Launch the CUDA Kernel
	dim3 block(N, N);
	dim3 grid(cols / N, rows / N);
	dtmKernel<<<grid, block>>>(reinterpret_cast<uint16*>(d_in), reinterpret_cast<uint16*>(d_out), minDist, maxDist,cols,rows);
	checkCudaErrors( hipGetLastError() );

	checkCudaErrors( hipEventRecord(stop, NULL) );
	checkCudaErrors( hipEventSynchronize(stop) );

	// Copy the device result to the host
	checkCudaErrors( hipMemcpy(h_out, d_out, SIZE, hipMemcpyDeviceToHost) );

	checkCudaErrors( hipFree(d_in) );
	checkCudaErrors( hipFree(d_out) );

	float msec = 0.f;
	checkCudaErrors( hipEventElapsedTime(&msec, start, stop) );

//#ifdef __CPU_VERSION__
	//printf("the gpu version takes %fl\n", msec);
//#endif
	return msec;
}

void dtmGpuColor( uint16* h_in, uchar* h_inCol,uchar* h_outCol, const int rows, const int cols, const int minDist, const int maxDist)
{
	const int SIZEU16 = rows * cols * sizeof(uint16);
	const int SIZEU8 = rows * cols * sizeof(uchar);
	hipEvent_t start, stop;
	
	checkCudaErrors( hipEventCreate(&start) );
	checkCudaErrors( hipEventCreate(&stop) );

	// Allocate the device input images
	uint16 *d_in = NULL;
	checkCudaErrors( hipMalloc((void **)&d_in, SIZEU16) );
	uchar *d_inCol = NULL;
	checkCudaErrors( hipMalloc((void **)&d_inCol, SIZEU8) );

	// Allocate the device output image
	uchar *d_out = NULL;
	checkCudaErrors( hipMalloc((void **)&d_out, SIZEU8) );

	// Copy the host input image  to the device memory
	checkCudaErrors( hipMemcpy(d_in, h_in, SIZEU16, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_inCol, h_inCol, SIZEU8, hipMemcpyHostToDevice) );

	checkCudaErrors( hipEventRecord(start, NULL) );

	// Launch the CUDA Kernel
	dim3 block(N, N);
	dim3 grid(cols / N, rows / N);
	//dtmKernelColor<<<grid, block>>>(d_in, d_inCol, d_out, minDist, maxDist,cols,rows);
	dtmKernelColorAlternative<<<grid, block>>>(d_in,d_inCol,minDist, maxDist,cols,rows);
	checkCudaErrors( hipGetLastError() );

	checkCudaErrors( hipEventRecord(stop, NULL) );
	checkCudaErrors( hipEventSynchronize(stop) );

	// Copy the device result to the host
	//checkCudaErrors( hipMemcpy(h_outCol, d_out, SIZEU8, hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy(h_outCol, d_in, SIZEU8, hipMemcpyDeviceToHost) );

	checkCudaErrors( hipFree(d_in) );
	checkCudaErrors( hipFree(d_inCol) );
	checkCudaErrors( hipFree(d_out) );

	float msec = 0.f;
	checkCudaErrors( hipEventElapsedTime(&msec, start, stop) );

	//printf("This took us %fl", msec);
}

void depthToRgbWorldPoint(uchar* h_in, float* h_transMat ,uchar* h_out, const int rows, const int cols, const int minDist, const int maxDist)
{
		const int SIZEU16 = rows * cols * sizeof(uint16);
		const int SIZEWP = rows * cols * sizeof(SWorldPoint);
		const int TRMATRC = 4;
		const int SIZETRMAT = TRMATRC*TRMATRC*sizeof(float);
		hipEvent_t start, stop;


		checkCudaErrors( hipEventCreate(&start) );
		checkCudaErrors( hipEventCreate(&stop) );

		// Allocate the device input images
		uchar *d_in = NULL;
		checkCudaErrors( hipMalloc((void **)&d_in, SIZEU16) );
		uchar *d_trMat = NULL;
		checkCudaErrors( hipMalloc((void **)&d_trMat, SIZETRMAT) );

		// Allocate the device output image
		uchar *d_out = NULL;
		checkCudaErrors( hipMalloc((void **)&d_out, SIZEWP) );

		// Copy the host input image  to the device memory
		checkCudaErrors( hipMemcpy(d_in, h_in, SIZEU16, hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(d_trMat, h_transMat, SIZETRMAT, hipMemcpyHostToDevice) );

		checkCudaErrors( hipEventRecord(start, NULL) );

		// Launch the CUDA Kernel
		dim3 block(N, N);
		dim3 grid(cols / N, rows / N);
		depthToWorldColorKernel<<<grid, block>>>(reinterpret_cast<uint16*>(d_in), reinterpret_cast<float*>(d_trMat), reinterpret_cast<SWorldPoint*>(d_out), minDist, maxDist,cols,rows);
		checkCudaErrors( hipGetLastError() );

		checkCudaErrors( hipEventRecord(stop, NULL) );
		checkCudaErrors( hipEventSynchronize(stop) );

		// Copy the device result to the host
		checkCudaErrors( hipMemcpy(h_out, d_out, SIZEWP, hipMemcpyDeviceToHost) );

		checkCudaErrors( hipFree(d_in) );
		checkCudaErrors( hipFree(d_trMat) );
		checkCudaErrors( hipFree(d_out) );

		float msec = 0.f;
		checkCudaErrors( hipEventElapsedTime(&msec, start, stop) );

		//printf("This took us %fl", msec);
	}


